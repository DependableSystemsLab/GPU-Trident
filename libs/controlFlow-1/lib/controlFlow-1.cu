#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <unistd.h>
#include "local_param.h"

__device__ float control_flow_rec[Y_MAX][X_MAX][1000];
__device__ int count[Y_MAX][X_MAX];

extern "C" __device__ void profileCmp(int cmpResult, long index)
{    
	float a = (float)cmpResult/10;
    
    if (index > START_LOOP && index < END_LOOP)
    {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;  
        int idy = blockIdx.y * blockDim.y + threadIdx.y;

        unsigned long long local_index = atomicAdd(&count[idy][idx],1);

        control_flow_rec[idy][idx][local_index] = (float)index + a;
    }
}

