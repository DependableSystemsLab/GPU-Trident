
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <unistd.h>

#define LIST_SIZE 100000
__device__ unsigned long long instCountList[LIST_SIZE];
__device__ unsigned long long record_flag = 0;


extern "C" __device__ void profileCount(long index){
    
    if (record_flag == 0)
        return;
    
    atomicAdd(&instCountList[index], 1);
}
