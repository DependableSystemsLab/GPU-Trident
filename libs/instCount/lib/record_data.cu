#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <hip/hip_runtime.h>





#define LIST_SIZE 100000
extern "C" __device__ long long instCountList[LIST_SIZE];
extern "C" __device__ unsigned long long record_flag;

void bambooLogRecordOff(){

    long long local_record = 0;

    hipMemcpyToSymbol(HIP_SYMBOL(record_flag), &local_record, sizeof(long long), 0, hipMemcpyHostToDevice);
}

void bambooLogKernelBegin(long long i) {

    i = 1;

    hipMemcpyToSymbol(HIP_SYMBOL(record_flag), &i, sizeof(long long), 0, hipMemcpyHostToDevice);
}

void bambooLogKernelEnd() {

#ifdef KERNELTRACE
	cudaDeviceSynchronize();
#endif
	
	long long resultArray[LIST_SIZE] = {0};
	hipMemcpyFromSymbol(&resultArray, HIP_SYMBOL(instCountList), LIST_SIZE * sizeof(long long), 0, hipMemcpyDeviceToHost);    
    
	FILE *profileFile = fopen("instCountResult.txt", "w");
	for(long long i=0; i<LIST_SIZE; i++){
		if(resultArray[i] != 0){
			fprintf(profileFile, "%lld: %lld\n", i, resultArray[i]);
		}
	}
	fclose(profileFile);
	memset(resultArray, 0, sizeof(resultArray));
	hipMemcpyToSymbol(HIP_SYMBOL(instCountList), &resultArray, LIST_SIZE * sizeof(long long), 0, hipMemcpyHostToDevice);
}
