#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <hip/hip_runtime.h>





#define LIST_SIZE 100000
extern "C" __device__ long long instCountList[LIST_SIZE];

void bambooLogKernelBegin() {

}

void bambooLogKernelEnd() {

#ifdef KERNELTRACE
	cudaDeviceSynchronize();
#endif
	
	long long resultArray[LIST_SIZE] = {0};
	hipMemcpyFromSymbol(&resultArray, HIP_SYMBOL(instCountList), LIST_SIZE * sizeof(long long), 0, hipMemcpyDeviceToHost);    
    
	FILE *profileFile = fopen("instCountResult.txt", "w");
	for(long long i=0; i<LIST_SIZE; i++){
		if(resultArray[i] != 0){
			fprintf(profileFile, "%lld: %lld\n", i, resultArray[i]);
		}
	}
	fclose(profileFile);
	memset(resultArray, 0, sizeof(resultArray));
	hipMemcpyToSymbol(HIP_SYMBOL(instCountList), &resultArray, LIST_SIZE * sizeof(long long), 0, hipMemcpyHostToDevice);
}