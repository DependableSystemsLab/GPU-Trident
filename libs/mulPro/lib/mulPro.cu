
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <unistd.h>

#define LIST_SIZE 100000
__device__ int init_flag = 0;
__device__ unsigned long long mulValue1List[LIST_SIZE];
__device__ unsigned long long mulValue2List[LIST_SIZE];
__device__ unsigned long long mulCountList[LIST_SIZE];
__device__ unsigned long long record_flag;

extern "C" __device__ void profileMulValue(long mulValue1,long mulValue2, long index)
{
    if (record_flag == 0)
        return;
        
    atomicAdd(&mulCountList[index],1);
    if (mulValue1 == 0)
    {
    	atomicAdd(&mulValue1List[index], 1);
    }

    if(mulValue2 == 0)
    {
    	atomicAdd(&mulValue2List[index], 1);
    }
}

extern "C" __device__ void profileFmulValue(double mulValue1,double mulValue2, long index)
{	
    if (record_flag == 0)
        return;
       
    atomicAdd(&mulCountList[index],1);
    
    if (mulValue1 == 0)
    {
    	atomicAdd(&mulValue1List[index], 1);
    }

    if (mulValue2 == 0)
    {
    	atomicAdd(&mulValue1List[index], 1);
    }
}

