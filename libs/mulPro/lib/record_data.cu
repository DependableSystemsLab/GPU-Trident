#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <hip/hip_runtime.h>





#define LIST_SIZE 100000
extern "C" __device__ unsigned long long mulValue1List[LIST_SIZE];
extern "C" __device__ unsigned long long mulValue2List[LIST_SIZE];
extern "C" __device__ unsigned long long mulCountList[LIST_SIZE];
extern "C" __device__ unsigned long long record_flag;

void bambooLogRecordOff(){

    long long local_record = 0;

    hipMemcpyToSymbol(HIP_SYMBOL(record_flag), &local_record, sizeof(long long), 0, hipMemcpyHostToDevice);
}

void bambooLogKernelBegin(long long i) {

    i = 1;

    hipMemcpyToSymbol(HIP_SYMBOL(record_flag), &i, sizeof(long long), 0, hipMemcpyHostToDevice);
}

void bambooLogKernelEnd() 
{

#ifdef KERNELTRACE
    cudaDeviceSynchronize();
#endif
    
    long long mulValue1ListLocal[LIST_SIZE];
    long long mulValue2ListLocal[LIST_SIZE];
    long long mulCountListLocal[LIST_SIZE];
    
    hipMemcpyFromSymbol(mulValue1ListLocal, HIP_SYMBOL(mulValue1List), LIST_SIZE * sizeof(long long), 0, hipMemcpyDeviceToHost);
    hipMemcpyFromSymbol(mulValue2ListLocal, HIP_SYMBOL(mulValue2List), LIST_SIZE * sizeof(long long), 0, hipMemcpyDeviceToHost);
    hipMemcpyFromSymbol(mulCountListLocal, HIP_SYMBOL(mulCountList), LIST_SIZE * sizeof(long long), 0, hipMemcpyDeviceToHost);
    
    FILE *profileFile = fopen("profile_mul_value_result.txt", "w");
    for(long long i=0; i < LIST_SIZE; i++){
        if(mulCountListLocal[i] != 0)
        {
            fprintf(profileFile, "%lld %lld %lld %lld\n", i, mulCountListLocal[i], mulValue1ListLocal[i], mulValue2ListLocal[i]);
        }
    }
    
    fclose(profileFile);
}
