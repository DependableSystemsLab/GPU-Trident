#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <hip/hip_runtime.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <>
#include "local_param.h"

extern "C" __device__ float control_flow_rec[Y_MAX][X_MAX][CF_2_NUM];
extern "C" __device__ int count[Y_MAX][X_MAX];

int lc;

void bambooLogKernelBegin(long long int invoc_count) {

    float controlflow[CF_2_NUM];
    int local_count[1] = {0};

    memset(controlflow, 0, sizeof(controlflow));
    
    for (long long i =0; i < Y_MAX; i++)
    {
        for (long long j =0; j < X_MAX; j++)
        {
            hipMemcpyToSymbol(HIP_SYMBOL(control_flow_rec), controlflow, CF_2_NUM * sizeof(float), j*CF_2_NUM*sizeof(float) + sizeof(float)*i*X_MAX*CF_2_NUM, hipMemcpyHostToDevice);
            hipMemcpyToSymbol(HIP_SYMBOL(count), local_count, sizeof(int), j* sizeof(int) + sizeof(int)*i*X_MAX, hipMemcpyHostToDevice);
        }
    }

    lc = invoc_count;
}

void bambooLogRecordOff()
{
    hipDeviceSynchronize();

    float controlflow[CF_2_NUM];

    memset(controlflow, 0, sizeof(controlflow));

    int temp;
    char cond_str;
    
    FILE *profileFile1 = fopen("control_flow_group-2.txt", "a");

    for (long long k = 0; k < Y_MAX; k++)
    {
        for (long long j =0; j < X_MAX; j++)
        {
            hipMemcpyFromSymbol(controlflow, HIP_SYMBOL(control_flow_rec), CF_2_NUM * sizeof(float), j*CF_2_NUM*sizeof(float) + sizeof(float)*k*X_MAX*CF_2_NUM, hipMemcpyDeviceToHost);

            fprintf(profileFile1, "%d %lld %lld:", lc, j, k);

            for (long long i = 0; i < CF_2_NUM && controlflow[i] != 0; i++)
            {
                temp = (int)floor(controlflow[i]);

                cond_str = (controlflow[i] > (float)temp) ? 'T' : 'F';

                fprintf(profileFile1, "%d%c", temp, cond_str);
            }

            fprintf(profileFile1, "\n");
        }
    }

    fclose(profileFile1);
}

void bambooLogKernelEnd()
{

}
