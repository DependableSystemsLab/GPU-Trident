#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <hip/hip_runtime.h>





#define LIST_SIZE 2000
#define LIST_SIZE_GLOBAL 500000

extern "C" __device__ unsigned long long load_store_index[LIST_SIZE];
extern "C" __device__ unsigned long long load_store_address[LIST_SIZE];
extern "C" __device__ unsigned long long load_store_check[LIST_SIZE];
extern "C" __device__ unsigned long long record_flag;
extern "C" __device__ int index_c;

void bambooLogRecordOff(){

    long long local_record = 0;

    hipMemcpyToSymbol(HIP_SYMBOL(record_flag), &local_record, sizeof(long long), 0, hipMemcpyHostToDevice);
}

void bambooLogKernelBegin(long long i) {

    i = 1;

    hipMemcpyToSymbol(HIP_SYMBOL(record_flag), &i, sizeof(long long), 0, hipMemcpyHostToDevice);
    
    hipMemcpyFromSymbol(&i, HIP_SYMBOL(index_c), sizeof(int), 0, hipMemcpyDeviceToHost);
    
    printf("How much:%lld\n\n", i);
}

void bambooLogKernelEnd()
{
    
    long long i = 0;
    
    hipMemcpyFromSymbol(&i, HIP_SYMBOL(index_c), sizeof(int), 0, hipMemcpyDeviceToHost);
    
    printf("How much End:%lld\n\n", i);
    
	hipDeviceSynchronize();

	unsigned long long loadStoreIndex[LIST_SIZE] = {0};
	unsigned long long loadStoreAddress[LIST_SIZE] = {0};
	unsigned long long loadStoreCheck[LIST_SIZE] = {0};
	
	FILE *profileFile = fopen("profile_mem_result.txt", "w");


    for (long long j = 0; j <  LIST_SIZE_GLOBAL; j += LIST_SIZE)
    {
    
	    hipMemcpyFromSymbol(loadStoreIndex, HIP_SYMBOL(load_store_index), LIST_SIZE * sizeof(long long), j*sizeof(long long), hipMemcpyDeviceToHost);
	    hipMemcpyFromSymbol(loadStoreAddress, HIP_SYMBOL(load_store_address), LIST_SIZE * sizeof(long long), j*sizeof(long long), hipMemcpyDeviceToHost);
	    hipMemcpyFromSymbol(loadStoreCheck, HIP_SYMBOL(load_store_check), LIST_SIZE * sizeof(long long), j*sizeof(long long), hipMemcpyDeviceToHost);
	
	
	    for(long long i=0; i < LIST_SIZE; i++)
	    {
            if(loadStoreIndex[i] != 0)
            {
                if (loadStoreCheck[i] == 0)
                {
                    fprintf(profileFile, "L %lld %p\n", loadStoreIndex[i], (void*)loadStoreAddress[i]);   
                }
                else 
                {   
                    fprintf(profileFile, "S %lld %p\n", loadStoreIndex[i], (void*)loadStoreAddress[i]);   
                }
		    }
	    }
	}
		
	fclose(profileFile);
/*
	memset(loadStoreIndex, 0, sizeof(loadStoreIndex));
	memset(loadStoreAddress, 0, sizeof(loadStoreAddress));
	memset(loadStoreCheck, 0, sizeof(loadStoreCheck));
	
	cudaMemcpyToSymbol(load_store_index, &loadStoreIndex, LIST_SIZE * sizeof(long long), 0, cudaMemcpyHostToDevice);
	cudaMemcpyToSymbol(load_store_address, &loadStoreAddress, LIST_SIZE * sizeof(long long), 0, cudaMemcpyHostToDevice);
	cudaMemcpyToSymbol(load_store_check, &loadStoreCheck, LIST_SIZE * sizeof(long long), 0, cudaMemcpyHostToDevice);
*/	
}
