#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <hip/hip_runtime.h>





#define LIST_SIZE 100000
extern "C" __device__ unsigned long long load_store_index[LIST_SIZE];
extern "C" __device__ unsigned long long load_store_address[LIST_SIZE];
extern "C" __device__ unsigned long long load_store_check[LIST_SIZE];

void bambooLogKernelBegin() {

}

void bambooLogKernelEnd()
{

#ifdef KERNELTRACE
	cudaDeviceSynchronize();
#endif

	unsigned long long loadStoreIndex[LIST_SIZE] = {0};
	unsigned long long loadStoreAddress[LIST_SIZE] = {0};
	unsigned long long loadStoreCheck[LIST_SIZE] = {0};
	
	hipMemcpyFromSymbol(&loadStoreIndex, HIP_SYMBOL(load_store_index), LIST_SIZE * sizeof(long long), 0, hipMemcpyDeviceToHost);
	hipMemcpyFromSymbol(&loadStoreAddress, HIP_SYMBOL(load_store_address), LIST_SIZE * sizeof(long long), 0, hipMemcpyDeviceToHost);
	hipMemcpyFromSymbol(&loadStoreCheck, HIP_SYMBOL(load_store_check), LIST_SIZE * sizeof(long long), 0, hipMemcpyDeviceToHost);
    
	FILE *profileFile = fopen("profile_mem_result.txt", "w");
	
	for(long long i=0; i < LIST_SIZE; i++){
        
        if(loadStoreIndex[i] != 0){
        
            if (loadStoreCheck[i] == 0)
            {
                fprintf(profileFile, "L %lld %p\n", loadStoreIndex[i], (void*)loadStoreAddress[i]);   
            }
            else 
            {   
                fprintf(profileFile, "S %lld %p\n", loadStoreIndex[i], (void*)loadStoreAddress[i]);   
            }
		}
	}
				
	fclose(profileFile);
	memset(loadStoreIndex, 0, sizeof(loadStoreIndex));
	memset(loadStoreAddress, 0, sizeof(loadStoreAddress));
	memset(loadStoreCheck, 0, sizeof(loadStoreCheck));
	
	hipMemcpyToSymbol(HIP_SYMBOL(load_store_index), &loadStoreIndex, LIST_SIZE * sizeof(long long), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(load_store_address), &loadStoreAddress, LIST_SIZE * sizeof(long long), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(load_store_check), &loadStoreCheck, LIST_SIZE * sizeof(long long), 0, hipMemcpyHostToDevice);
}
