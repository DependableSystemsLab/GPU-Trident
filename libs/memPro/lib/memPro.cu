
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <unistd.h>

#define LIST_SIZE 500000
__device__ int index_c = 0;
__device__ unsigned long long load_store_index[LIST_SIZE];
__device__ unsigned long long load_store_address[LIST_SIZE];
__device__ unsigned long long load_store_check[LIST_SIZE];
__device__ unsigned long long record_flag = 0;


extern "C" __device__ void profileLoadInst(long* adress, long index){

    //int idx = blockIdx.x * blockDim.x + threadIdx.x;  
    //int idy = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (record_flag != 0)
    {
        unsigned long long local_index = atomicAdd(&index_c,1);
        atomicAdd(&load_store_address[local_index], (long)adress);
        atomicAdd(&load_store_index[local_index], index);
        atomicAdd(&load_store_check[local_index], 0);
    }    
}

extern "C" __device__ void profileStoreInst(long* adress, long index){

    //int idx = blockIdx.x * blockDim.x + threadIdx.x;  
    //int idy = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (record_flag != 0)
    {
        unsigned long long local_index = atomicAdd(&index_c,1);
        atomicAdd(&load_store_address[local_index], (long)adress);
        atomicAdd(&load_store_index[local_index], index);
        atomicAdd(&load_store_check[local_index], 1);
    }
}

