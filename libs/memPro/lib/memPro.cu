
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <unistd.h>

#define LIST_SIZE 100000
__device__ int index_c = 0;
__device__ unsigned long long load_store_index[LIST_SIZE];
__device__ unsigned long long load_store_address[LIST_SIZE];
__device__ unsigned long long load_store_check[LIST_SIZE];


extern "C" __device__ void profileLoadInst(long* adress, long index){

    unsigned long long local_index = atomicAdd(&index_c,1);
    atomicAdd(&load_store_address[local_index], (long)adress);
    atomicAdd(&load_store_index[local_index], index);
    atomicAdd(&load_store_check[local_index], 0);
    
/*	
	if(init_flag == 0){
		int i = 0;
		for(i=0;i<LIST_SIZE;i++){
			oneList[i] = 0;
			zeroList[i] = 0;
		}
		atomicAdd(&init_flag, 1);
	}
*/
    
}

extern "C" __device__ void profileStoreInst(long* adress, long index){

    unsigned long long local_index = atomicAdd(&index_c,1);
    atomicAdd(&load_store_address[local_index], (long)adress);
    atomicAdd(&load_store_index[local_index], index);
    atomicAdd(&load_store_check[local_index], 1);
/*	
	if(init_flag == 0){
		int i = 0;
		for(i=0;i<LIST_SIZE;i++){
			oneList[i] = 0;
			zeroList[i] = 0;
		}
		atomicAdd(&init_flag, 1);
	}
*/
    
}

