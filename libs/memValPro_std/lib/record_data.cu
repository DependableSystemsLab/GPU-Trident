#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <hip/hip_runtime.h>





#define LIST_SIZE_GLOBAL 5000000
#define LIST_SIZE 10000
extern "C" __device__ unsigned  long long load_store_index[LIST_SIZE];
extern "C" __device__ unsigned long long load_store_value[LIST_SIZE];
extern "C" __device__ double load_store_double[LIST_SIZE];
extern "C" __device__ unsigned long long load_store_double_index[LIST_SIZE];
extern "C" __device__ unsigned long long record_flag;
extern "C" __device__ unsigned long long call_count;

int memPro_kernel = 0;

void bambooLogRecordOff(){

    long long local_record = 0;

    hipMemcpyToSymbol(HIP_SYMBOL(record_flag), &local_record, sizeof(long long), 0, hipMemcpyHostToDevice);
}


void bambooLogKernelBegin(long long i) {

    hipMemcpyToSymbol(HIP_SYMBOL(call_count), &i, sizeof(long long), 0, hipMemcpyHostToDevice);

    i = 1;

    hipMemcpyToSymbol(HIP_SYMBOL(record_flag), &i, sizeof(long long), 0, hipMemcpyHostToDevice);
}

void bambooLogKernelEnd()
{

    unsigned long long loadStoreIndex[LIST_SIZE] = {0};
    unsigned long long loadStoreValue[LIST_SIZE] = {0};

    unsigned long long loadStoreIndex_double[LIST_SIZE] = {0};
    double loadStoreValue_double[LIST_SIZE] = {0};

    FILE *profileFile = fopen("profile_mem_val_result.txt", "a");

    
    for (int j=0; j < LIST_SIZE_GLOBAL; j+=LIST_SIZE)
    {
        hipMemcpyFromSymbol(&loadStoreIndex, HIP_SYMBOL(load_store_index), LIST_SIZE * sizeof(unsigned long long), j*sizeof(unsigned long long), hipMemcpyDeviceToHost);
        hipMemcpyFromSymbol(&loadStoreValue, HIP_SYMBOL(load_store_value), LIST_SIZE * sizeof(unsigned long long), j*sizeof(unsigned long long), hipMemcpyDeviceToHost);
    
        for(long long i=0; i < LIST_SIZE && loadStoreIndex[i] != 0; i++)
        {

           fprintf(profileFile, "%lld %lld\n", loadStoreIndex[i], loadStoreValue[i]);   
        }
    }
    
    for (int j=0; j < LIST_SIZE_GLOBAL; j+=LIST_SIZE)
    {
        hipMemcpyFromSymbol(&loadStoreIndex_double, HIP_SYMBOL(load_store_double_index), LIST_SIZE * sizeof(unsigned long long), j*sizeof(unsigned long long), hipMemcpyDeviceToHost);
        hipMemcpyFromSymbol(&loadStoreValue_double, HIP_SYMBOL(load_store_double), LIST_SIZE * sizeof(double), j*sizeof(double), hipMemcpyDeviceToHost);
    
        for(long long i=0; i < LIST_SIZE && loadStoreIndex_double[i] != 0; i++)
        {

           fprintf(profileFile, "%lld %.40f\n", loadStoreIndex_double[i], loadStoreValue_double[i]);   
        }
    }
    
    fclose(profileFile);
}
