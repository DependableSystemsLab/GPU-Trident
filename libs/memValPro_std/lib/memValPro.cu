#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <unistd.h>

#define INDEX
#define LIST_SIZE 5000000
__device__ int index_int = 0;
__device__ int index_float = 0;

__device__ unsigned long long load_store_index[LIST_SIZE] = {0};
__device__ unsigned long long load_store_value[LIST_SIZE] = {0};
__device__ double load_store_double[LIST_SIZE] = {0.0};
__device__ unsigned long long load_store_double_index[LIST_SIZE] = {0};
__device__ unsigned long long record_flag;
__device__ unsigned long long call_count;

extern "C" __device__ void profileStoreInst(long value, long index){

    if (record_flag == 0)
        return;

    if (INDEX)
    {
        long local_index = atomicAdd(&index_int,1);
        load_store_value[local_index] = (unsigned long long)value;
        load_store_index[local_index] = (unsigned long long)index;
    }
}

extern "C" __device__ void profileStoreInstfloat(double value, long index){

    if (record_flag == 0)
        return;

    if (INDEX)
    {
        long local_index = atomicAdd(&index_float,1);
        load_store_double[local_index] = value;
        load_store_double_index[local_index] = index;
    } 
}

