#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <hip/hip_runtime.h>





#define LIST_SIZE 100000
extern "C" __device__ unsigned long long shiftCount[LIST_SIZE];
extern "C" __device__ unsigned long long shiftVal[LIST_SIZE];

void bambooLogKernelBegin() {

}

void bambooLogKernelEnd()
{

#ifdef KERNELTRACE
	cudaDeviceSynchronize();
#endif

	unsigned long long shift_count[LIST_SIZE] = {0};
	unsigned long long shift_val[LIST_SIZE] = {0};
	
	hipMemcpyFromSymbol(&shift_count, HIP_SYMBOL(shiftCount), LIST_SIZE * sizeof(long long), 0, hipMemcpyDeviceToHost);
	hipMemcpyFromSymbol(&shift_val, HIP_SYMBOL(shiftVal), LIST_SIZE * sizeof(long long), 0, hipMemcpyDeviceToHost);
    
	FILE *profileFile = fopen("profile_shift_value_result.txt", "w");
	
	for(long long i=0; i < LIST_SIZE; i++){
        
        if(shift_count[i] != 0){
		    fprintf(profileFile, "%lld: %lld %lld\n", i, shift_val[i]/shift_count[i], shift_count[i]);
		}
	}
				
	fclose(profileFile);
	memset(shift_count, 0, sizeof(shift_count));
	memset(shift_val, 0, sizeof(shift_val));
	hipMemcpyToSymbol(HIP_SYMBOL(shiftCount), &shift_count, LIST_SIZE * sizeof(long long), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(shiftVal), &shift_val, LIST_SIZE * sizeof(long long), 0, hipMemcpyHostToDevice);
}