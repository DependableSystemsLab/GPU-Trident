#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <hip/hip_runtime.h>





#define LIST_SIZE 100000
extern "C" __device__ unsigned long long shiftCount[LIST_SIZE];
extern "C" __device__ unsigned long long shiftVal[LIST_SIZE];
extern "C" __device__ unsigned long long record_flag;

void bambooLogRecordOff(){

    long long local_record = 0;

    hipMemcpyToSymbol(HIP_SYMBOL(record_flag), &local_record, sizeof(long long), 0, hipMemcpyHostToDevice);
}

void bambooLogKernelBegin(long long i) {

    i = 1;

    hipMemcpyToSymbol(HIP_SYMBOL(record_flag), &i, sizeof(long long), 0, hipMemcpyHostToDevice);
}

void bambooLogKernelEnd()
{

#ifdef KERNELTRACE
    cudaDeviceSynchronize();
#endif

    unsigned long long shift_count[LIST_SIZE] = {0};
    unsigned long long shift_val[LIST_SIZE] = {0};
    
    hipMemcpyFromSymbol(shift_count, HIP_SYMBOL(shiftCount), LIST_SIZE * sizeof(long long), 0, hipMemcpyDeviceToHost);
    hipMemcpyFromSymbol(shift_val, HIP_SYMBOL(shiftVal), LIST_SIZE * sizeof(long long), 0, hipMemcpyDeviceToHost);
    
    FILE *profileFile = fopen("profile_shift_value_result.txt", "w");
    
    for(long long i=0; i < LIST_SIZE; i++){
        
        if(shift_count[i] != 0){
            fprintf(profileFile, "%lld: %lld %lld\n", i, shift_val[i]/shift_count[i], shift_count[i]);
        }
    }
                
    fclose(profileFile);
}
