
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <unistd.h>

#define LIST_SIZE 100000
__device__ int init_flag = 0;
__device__ unsigned long long shiftCount[LIST_SIZE];
__device__ unsigned long long shiftVal[LIST_SIZE];


extern "C" __device__ void profileCmp(long shiftValue, long index){

    atomicAdd(&shiftCount[index],1);
    atomicAdd(&shiftCount[index],shiftValue);
/*	
	if(init_flag == 0){
		int i = 0;
		for(i=0;i<LIST_SIZE;i++){
			oneList[i] = 0;
			zeroList[i] = 0;
		}
		atomicAdd(&init_flag, 1);
	}
*/
    
}

