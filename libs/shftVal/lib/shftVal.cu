
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <unistd.h>

#define LIST_SIZE 100000
__device__ int init_flag = 0;
__device__ unsigned long long shiftCount[LIST_SIZE];
__device__ unsigned long long shiftVal[LIST_SIZE];
__device__ unsigned long long record_flag = 0;


extern "C" __device__ void profileShiftValues(long shiftValue, long index){

    if (record_flag != 0)
    {
        atomicAdd(&shiftCount[index],1);
        atomicAdd(&shiftCount[index],shiftValue);
    }
    
}

