#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <unistd.h>

#define LIST_SIZE 10000
__device__ int init_flag = 0;
__device__ unsigned long long icmpValue1List[LIST_SIZE];
__device__ unsigned long long icmpValue2List[LIST_SIZE];
__device__ double fcmpValue1List[LIST_SIZE];
__device__ double fcmpValue2List[LIST_SIZE];
__device__ unsigned long long icmpCountList[LIST_SIZE];
__device__ unsigned long long fcmpCountList[LIST_SIZE];
__device__ unsigned long long record_flag;

/* Overloading the atomic add function for CUDA, as it is not available for computer capability < 6.0.0 */
#if __CUDA_ARCH__ < 600
__device__ double atomicAdd(double* address, double val)
{
    unsigned long long int* address_as_ull =
                              (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;

    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
                        __double_as_longlong(val +
                               __longlong_as_double(assumed)));

    // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
    } while (assumed != old);

    return __longlong_as_double(old);
}
#endif

extern "C" __device__ void profileICmpValue(long cmpValue1,long cmpValue2, long index)
{
    if (record_flag == 0)
        return;
        
    atomicAdd(&icmpCountList[index],1);
    atomicAdd(&icmpValue1List[index], cmpValue1);
    atomicAdd(&icmpValue2List[index], cmpValue2);
}

extern "C" __device__ void profileFCmpValue(double cmpValue1,double cmpValue2, long index)
{
    if (record_flag == 0)
        return;
        
    atomicAdd(&fcmpCountList[index],1);
    atomicAdd(&fcmpValue1List[index], cmpValue1);
    atomicAdd(&fcmpValue2List[index], cmpValue2);
}

