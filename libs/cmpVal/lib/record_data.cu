#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <hip/hip_runtime.h>





#define LIST_SIZE 10000
extern "C" __device__ unsigned long long icmpValue1List[LIST_SIZE];
extern "C" __device__ unsigned long long icmpValue2List[LIST_SIZE];
extern "C" __device__ double fcmpValue1List[LIST_SIZE];
extern "C" __device__ double fcmpValue2List[LIST_SIZE];
extern "C" __device__ unsigned long long icmpCountList[LIST_SIZE];
extern "C" __device__ unsigned long long fcmpCountList[LIST_SIZE];
extern "C" __device__ unsigned long long record_flag;

void bambooLogRecordOff(){

    long long local_record = 0;

    hipMemcpyToSymbol(HIP_SYMBOL(record_flag), &local_record, sizeof(long long), 0, hipMemcpyHostToDevice);
}

void bambooLogKernelBegin(long long i) {

    i = 1;

    hipMemcpyToSymbol(HIP_SYMBOL(record_flag), &i, sizeof(long long), 0, hipMemcpyHostToDevice);
}

void bambooLogKernelEnd() 
{

#ifdef KERNELTRACE
    cudaDeviceSynchronize();
#endif
    
    long long icmpValue1ListLocal[LIST_SIZE];
    long long icmpValue2ListLocal[LIST_SIZE];
    double fcmpValue1ListLocal[LIST_SIZE];
    double fcmpValue2ListLocal[LIST_SIZE];
    long long icmpCountListLocal[LIST_SIZE];
    long long fcmpCountListLocal[LIST_SIZE];
    
    hipMemcpyFromSymbol(icmpValue1ListLocal, HIP_SYMBOL(icmpValue1List), LIST_SIZE * sizeof(long long), 0, hipMemcpyDeviceToHost);
    hipMemcpyFromSymbol(icmpValue2ListLocal, HIP_SYMBOL(icmpValue2List), LIST_SIZE * sizeof(long long), 0, hipMemcpyDeviceToHost);
    hipMemcpyFromSymbol(fcmpValue1ListLocal, HIP_SYMBOL(fcmpValue1List), LIST_SIZE * sizeof(double), 0, hipMemcpyDeviceToHost);
    hipMemcpyFromSymbol(fcmpValue2ListLocal, HIP_SYMBOL(fcmpValue2List), LIST_SIZE * sizeof(double), 0, hipMemcpyDeviceToHost);
    hipMemcpyFromSymbol(icmpCountListLocal, HIP_SYMBOL(icmpCountList), LIST_SIZE * sizeof(long long), 0, hipMemcpyDeviceToHost);
    hipMemcpyFromSymbol(fcmpCountListLocal, HIP_SYMBOL(fcmpCountList), LIST_SIZE * sizeof(long long), 0, hipMemcpyDeviceToHost);
    
    FILE *profileFile = fopen("profile_cmp_value_result.txt", "w");
    for(long long i=0; i < LIST_SIZE; i++){
        if(icmpCountListLocal[i] != 0)
        {
            fprintf(profileFile, "icmp %lld: %lld %lld %lld\n", i, icmpValue1ListLocal[i]/icmpCountListLocal[i], icmpValue2ListLocal[i]/icmpCountListLocal[i], icmpCountListLocal[i]);
        }
        else if(fcmpCountListLocal[i] != 0)
        {
            fprintf(profileFile, "fcmp %lld: %f %f %lld\n", i, fcmpValue1ListLocal[i]/fcmpCountListLocal[i], fcmpValue2ListLocal[i]/fcmpCountListLocal[i], fcmpCountListLocal[i]);
        }
    }
    
    fclose(profileFile);
}
