#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <hip/hip_runtime.h>





#define LIST_SIZE 100000
extern "C" __device__ unsigned long long icmpValue1List[LIST_SIZE];
extern "C" __device__ unsigned long long icmpValue2List[LIST_SIZE];
extern "C" __device__ double fcmpValue1List[LIST_SIZE];
extern "C" __device__ double fcmpValue2List[LIST_SIZE];
extern "C" __device__ unsigned long long icmpCountList[LIST_SIZE];
extern "C" __device__ unsigned long long fcmpCountList[LIST_SIZE];

void bambooLogKernelBegin() {

}

void bambooLogKernelEnd() 
{

#ifdef KERNELTRACE
	cudaDeviceSynchronize();
#endif
	
	unsigned long long icmpValue1ListLocal[LIST_SIZE];
    unsigned long long icmpValue2ListLocal[LIST_SIZE];
    double fcmpValue1ListLocal[LIST_SIZE];
    double fcmpValue2ListLocal[LIST_SIZE];
    unsigned long long icmpCountListLocal[LIST_SIZE];
    unsigned long long fcmpCountListLocal[LIST_SIZE];
	
	hipMemcpyFromSymbol(&icmpValue1ListLocal, HIP_SYMBOL(icmpValue1List), LIST_SIZE * sizeof(long long), 0, hipMemcpyDeviceToHost);
	hipMemcpyFromSymbol(&icmpValue2ListLocal, HIP_SYMBOL(icmpValue2List), LIST_SIZE * sizeof(long long), 0, hipMemcpyDeviceToHost);
	hipMemcpyFromSymbol(&fcmpValue1ListLocal, HIP_SYMBOL(fcmpValue1List), LIST_SIZE * sizeof(double), 0, hipMemcpyDeviceToHost);
	hipMemcpyFromSymbol(&fcmpValue2ListLocal, HIP_SYMBOL(fcmpValue2List), LIST_SIZE * sizeof(double), 0, hipMemcpyDeviceToHost);
	hipMemcpyFromSymbol(&icmpCountListLocal, HIP_SYMBOL(icmpCountList), LIST_SIZE * sizeof(long long), 0, hipMemcpyDeviceToHost);
	hipMemcpyFromSymbol(&fcmpCountListLocal, HIP_SYMBOL(fcmpCountList), LIST_SIZE * sizeof(long long), 0, hipMemcpyDeviceToHost);
    
	FILE *profileFile = fopen("profile_cmp_value_result.txt", "w");
	for(long long i=0; i < LIST_SIZE; i++){
		if(icmpCountListLocal[i] != 0)
		{
			fprintf(profileFile, "icmp %lld: %lld %lld %lld\n", i, icmpValue1ListLocal[i]/icmpCountListLocal[i], icmpValue2ListLocal[i]/icmpCountListLocal[i], icmpCountListLocal[i]);
		}
		else if(fcmpCountListLocal[i] != 0)
		{
			fprintf(profileFile, "fcmp %lld: %f %f %lld\n", i, fcmpValue1ListLocal[i]/fcmpCountListLocal[i], fcmpValue2ListLocal[i]/fcmpCountListLocal[i], fcmpCountListLocal[i]);
		}
	}
	
	fclose(profileFile);
	memset(icmpValue1ListLocal, 0, sizeof(icmpValue1ListLocal));
	memset(icmpValue2ListLocal, 0, sizeof(icmpValue2ListLocal));
	memset(fcmpValue1ListLocal, 0, sizeof(icmpValue2ListLocal));
	memset(fcmpValue2ListLocal, 0, sizeof(fcmpValue2ListLocal));
	memset(icmpCountListLocal, 0, sizeof(icmpCountListLocal));
	memset(fcmpCountListLocal, 0, sizeof(fcmpCountListLocal));
	
	hipMemcpyFromSymbol(&icmpValue1List, HIP_SYMBOL(icmpValue1ListLocal), LIST_SIZE * sizeof(long long), 0, hipMemcpyHostToDevice);
	hipMemcpyFromSymbol(&icmpValue2List, HIP_SYMBOL(icmpValue2ListLocal), LIST_SIZE * sizeof(long long), 0, hipMemcpyHostToDevice);
	hipMemcpyFromSymbol(&fcmpValue1List, HIP_SYMBOL(fcmpValue1ListLocal), LIST_SIZE * sizeof(double), 0, hipMemcpyHostToDevice);
	hipMemcpyFromSymbol(&fcmpValue2List, HIP_SYMBOL(fcmpValue2ListLocal), LIST_SIZE * sizeof(double), 0, hipMemcpyHostToDevice);
	hipMemcpyFromSymbol(&icmpCountList, HIP_SYMBOL(icmpCountListLocal), LIST_SIZE * sizeof(long long), 0, hipMemcpyHostToDevice);
	hipMemcpyFromSymbol(&fcmpCountList, HIP_SYMBOL(fcmpCountListLocal), LIST_SIZE * sizeof(long long), 0, hipMemcpyHostToDevice);
}