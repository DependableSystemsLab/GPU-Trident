#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <unistd.h>

#define INDEX
#define LIST_SIZE 3000000
__device__ int index_c = 0;
__device__ unsigned long long load_store_index[LIST_SIZE];
__device__ unsigned long long load_store_address[LIST_SIZE];
__device__ unsigned long long load_store_check[LIST_SIZE];
__device__ unsigned long long record_flag;
__device__ unsigned long long call_count;

#define BX blockIdx.x
#define BY blockIdx.y
#define TX threadIdx.x
#define TY threadIdx.y
#define DX blockDim.x
#define DY blockDim.y



extern "C" __device__ void profileLoadInst(long* adress, long index){

    if (record_flag == 0)
        return;

    if (LOAD)
    {
        if (COND)
        {
            unsigned long long local_index = atomicAdd(&index_c,1);
            atomicAdd(&load_store_address[local_index], (long)adress);
            atomicAdd(&load_store_index[local_index], index);
            atomicAdd(&load_store_check[local_index], 0);   
        }
    }
}

extern "C" __device__ void profileStoreInst(long* adress, long index){
    
    if (record_flag == 0)
        return;
    
    if (STORE)
    {
        if (COND)
        {
            unsigned long long local_index = atomicAdd(&index_c,1);
            atomicAdd(&load_store_address[local_index], (long)adress);
            atomicAdd(&load_store_index[local_index], index);
            atomicAdd(&load_store_check[local_index], 1);
        }

    }
}

