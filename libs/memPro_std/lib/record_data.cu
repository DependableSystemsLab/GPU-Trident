#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <hip/hip_runtime.h>





#define LIST_SIZE_GLOBAL 3000000
#define LIST_SIZE 10000
extern "C" __device__ unsigned  long long load_store_index[LIST_SIZE];
extern "C" __device__ unsigned long long load_store_address[LIST_SIZE];
extern "C" __device__ unsigned long long load_store_check[LIST_SIZE];
extern "C" __device__ unsigned long long record_flag;
extern "C" __device__ unsigned long long call_count;

int memPro_kernel = 0;

void bambooLogRecordOff(){

    long long local_record = 0;

    hipMemcpyToSymbol(HIP_SYMBOL(record_flag), &local_record, sizeof(long long), 0, hipMemcpyHostToDevice);
}


void bambooLogKernelBegin(long long i) {

    hipMemcpyToSymbol(HIP_SYMBOL(call_count), &i, sizeof(long long), 0, hipMemcpyHostToDevice);

    i = 1;

    hipMemcpyToSymbol(HIP_SYMBOL(record_flag), &i, sizeof(long long), 0, hipMemcpyHostToDevice);
}

void bambooLogKernelEnd()
{

    unsigned long long loadStoreIndex[LIST_SIZE] = {0};
    unsigned long long loadStoreAddress[LIST_SIZE] = {0};
    unsigned long long loadStoreCheck[LIST_SIZE] = {0};

    FILE *profileFile = fopen("profile_mem_result.txt", "a");

    for (int j=0; j < LIST_SIZE_GLOBAL; j+=LIST_SIZE)
    {
        hipMemcpyFromSymbol(loadStoreIndex, HIP_SYMBOL(load_store_index), LIST_SIZE * sizeof(long long), j*sizeof(long long), hipMemcpyDeviceToHost);
        hipMemcpyFromSymbol(loadStoreAddress, HIP_SYMBOL(load_store_address), LIST_SIZE * sizeof(long long), j*sizeof(long long), hipMemcpyDeviceToHost);
        hipMemcpyFromSymbol(loadStoreCheck, HIP_SYMBOL(load_store_check), LIST_SIZE * sizeof(long long), j*sizeof(long long), hipMemcpyDeviceToHost);
    
        for(long long i=0; i < LIST_SIZE; i++)
        {
            if(loadStoreIndex[i] != 0)
            {
        
                if (loadStoreCheck[i] == 0)
                {
                    fprintf(profileFile, "L %lld %p\n", loadStoreIndex[i], (void*)loadStoreAddress[i]);   
                }
                else 
                {   
                    fprintf(profileFile, "S %lld %p\n", loadStoreIndex[i], (void*)loadStoreAddress[i]);   
                }
            }
        }
    }

    fclose(profileFile);
}
