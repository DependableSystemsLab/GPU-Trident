#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <unistd.h>

#define INDEX
#define LIST_SIZE 3000000
__device__ int index_c = 0;
__device__ unsigned long long load_store_index[LIST_SIZE];
__device__ unsigned long long load_store_address[LIST_SIZE];
__device__ unsigned long long load_store_check[LIST_SIZE];
__device__ unsigned long long record_flag;
__device__ unsigned long long call_count;



extern "C" __device__ void profileLoadInst(long* adress, long index){

    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int idy = threadIdx.y + blockIdx.y * blockDim.y;

    if (record_flag == 0)
        return;

    if (COND)
    {
        unsigned long long local_index = atomicAdd(&index_c,1);
        atomicAdd(&load_store_address[local_index], (long)adress);
        atomicAdd(&load_store_index[local_index], index);
        atomicAdd(&load_store_check[local_index], 0);   
    }
}

extern "C" __device__ void profileStoreInst(long* adress, long index){
    
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int idy = threadIdx.y + blockIdx.y * blockDim.y;

    if (record_flag == 0)
        return;
    
    if (COND)
    {
        unsigned long long local_index = atomicAdd(&index_c,1);
        atomicAdd(&load_store_address[local_index], (long)adress);
        atomicAdd(&load_store_index[local_index], index);
        atomicAdd(&load_store_check[local_index], 1);
    }
}

