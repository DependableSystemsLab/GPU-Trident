
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <unistd.h>

#define LIST_SIZE 100000
__device__ int init_flag = 0;
__device__ unsigned long long zeroList[LIST_SIZE];
__device__ unsigned long long oneList[LIST_SIZE];


extern "C" __device__ void profileCmp(int cmpResult, long index){
	
	if(init_flag == 0){
		int i = 0;
		for(i=0;i<LIST_SIZE;i++){
			oneList[i] = 0;
			zeroList[i] = 0;
		}
		atomicAdd(&init_flag, 1);
	}

    if (cmpResult == 0)
    {
        atomicAdd(&zeroList[index],1);
    }
    else
    {
        atomicAdd(&oneList[index],1);
    }
}

