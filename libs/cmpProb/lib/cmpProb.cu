
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <unistd.h>

#define LIST_SIZE 100000
__device__ unsigned long long zeroList[LIST_SIZE];
__device__ unsigned long long oneList[LIST_SIZE];
__device__ unsigned long long record_flag = 0;


extern "C" __device__ void profileCmp(int cmpResult, long index){
	
	if (record_flag == 0)
	    return;

    if (cmpResult == 0)
    {
        atomicAdd(&zeroList[index],1);
    }
    else
    {
        atomicAdd(&oneList[index],1);
    }
}

