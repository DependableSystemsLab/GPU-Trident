#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <hip/hip_runtime.h>





#define LIST_SIZE 100000
extern "C" __device__ unsigned long long zeroList[LIST_SIZE];
extern "C" __device__ unsigned long long oneList[LIST_SIZE];
extern "C" __device__ unsigned long long record_flag;

void bambooLogRecordOff(){

    long long local_record = 0;

    hipMemcpyToSymbol(HIP_SYMBOL(record_flag), &local_record, sizeof(long long), 0, hipMemcpyHostToDevice);
}

void bambooLogKernelBegin(long long i) {

    i = 1;

    hipMemcpyToSymbol(HIP_SYMBOL(record_flag), &i, sizeof(long long), 0, hipMemcpyHostToDevice);
}

void bambooLogKernelEnd() 
{

#ifdef KERNELTRACE
    cudaDeviceSynchronize();
#endif

    unsigned long long zero_result[LIST_SIZE] = {0};
    unsigned long long one_result[LIST_SIZE] = {0};
    
    hipMemcpyFromSymbol(zero_result, HIP_SYMBOL(zeroList), LIST_SIZE * sizeof(long long), 0, hipMemcpyDeviceToHost);
    hipMemcpyFromSymbol(one_result, HIP_SYMBOL(oneList), LIST_SIZE * sizeof(long long), 0, hipMemcpyDeviceToHost);
    
    FILE *profileFile = fopen("profile_cmp_prob_result.txt", "w");
    for(long long i=0; i < LIST_SIZE; i++){
        if(zero_result[i] != 0 || one_result[i] != 0){
            fprintf(profileFile, "%lld: %lld %lld\n", i, zero_result[i], one_result[i]);
        }
    }
    
    fclose(profileFile);
}
