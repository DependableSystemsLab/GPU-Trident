
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <unistd.h>

#define LIST_SIZE 100000
__device__ unsigned long long callCountList[LIST_SIZE];
__device__ int init_flag = 0;


extern "C" __device__ void callCount(long index){
    
	if(init_flag == 0){
		int i = 0;
		for(i=0;i<LIST_SIZE;i++){
			callCountList[i] = 0;
		}
		//init_flag = 1;
		atomicAdd(&init_flag, 1);
	}
    
    atomicAdd(&callCountList[index], 1);
}
