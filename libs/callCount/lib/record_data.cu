#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <hip/hip_runtime.h>





#define LIST_SIZE 100000
extern "C" __device__ long long callCountList[LIST_SIZE];

void bambooLogKernelBegin() {

}

void bambooLogRecordOff() {

}

void bambooLogKernelEnd() {

#ifdef KERNELTRACE
    cudaDeviceSynchronize();
#endif
    
    long long resultArray[LIST_SIZE] = {0};
    hipMemcpyFromSymbol(&resultArray, HIP_SYMBOL(callCountList), LIST_SIZE * sizeof(long long), 0, hipMemcpyDeviceToHost);    
    
    FILE *profileFile = fopen("profile_call_prob_result.txt", "w");
    for(long long i=0; i<LIST_SIZE; i++){
        if(resultArray[i] != 0){
            fprintf(profileFile, "%lld: %lld\n", i, resultArray[i]);
        }
    }
    fclose(profileFile);
}
