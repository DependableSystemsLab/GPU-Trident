#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <hip/hip_runtime.h>





#define LIST_SIZE_GLOBAL 5000000
#define LIST_SIZE 10000
extern "C" __device__ unsigned int load_store_index[LIST_SIZE];
extern "C" __device__ unsigned long load_store_value[LIST_SIZE];
extern "C" __device__ double load_store_double[LIST_SIZE];
extern "C" __device__ unsigned int load_store_double_index[LIST_SIZE];
extern "C" __device__ unsigned long long record_flag;
extern "C" __device__ unsigned long long call_count;

extern "C" __device__ int index_float;

int memPro_kernel = 0;

void bambooLogRecordOff(){

    long long local_record = 0;

    hipMemcpyToSymbol(HIP_SYMBOL(record_flag), &local_record, sizeof(long long), 0, hipMemcpyHostToDevice);
}


void bambooLogKernelBegin(long long i) {

    hipMemcpyToSymbol(HIP_SYMBOL(call_count), &i, sizeof(long long), 0, hipMemcpyHostToDevice);

    i = 1;

    hipMemcpyToSymbol(HIP_SYMBOL(record_flag), &i, sizeof(long long), 0, hipMemcpyHostToDevice);
}

void bambooLogKernelEnd()
{

    unsigned int loadStoreIndex[LIST_SIZE] = {0};
    unsigned long loadStoreValue[LIST_SIZE] = {0};

    unsigned int loadStoreIndex_double[LIST_SIZE] = {0};
    double loadStoreValue_double[LIST_SIZE] = {0};

    int index_float_local = 0;

    hipMemcpyFromSymbol(&index_float_local, HIP_SYMBOL(index_float), sizeof(int),0, hipMemcpyDeviceToHost);

    printf("Num elements:%d\n\n", index_float_local);


    FILE *profileFile = fopen("profile_mem_val_result.txt", "a");

    
    for (int j=0; j < LIST_SIZE_GLOBAL; j+=LIST_SIZE)
    {
        hipMemcpyFromSymbol(&loadStoreIndex, HIP_SYMBOL(load_store_index), LIST_SIZE * sizeof(unsigned int), j*sizeof(unsigned int), hipMemcpyDeviceToHost);
        hipMemcpyFromSymbol(&loadStoreValue, HIP_SYMBOL(load_store_value), LIST_SIZE * sizeof(unsigned long), j*sizeof(unsigned long), hipMemcpyDeviceToHost);
    
        for(long long i=0; i < LIST_SIZE && loadStoreIndex[i] != 0; i++)
        {

           fprintf(profileFile, "%u %ld\n", loadStoreIndex[i], loadStoreValue[i]);   
        }
    }
    
    for (int j=0; j < LIST_SIZE_GLOBAL; j+=LIST_SIZE)
    {
        hipMemcpyFromSymbol(&loadStoreIndex_double, HIP_SYMBOL(load_store_double_index), LIST_SIZE * sizeof(unsigned int), j*sizeof(unsigned int), hipMemcpyDeviceToHost);
        hipMemcpyFromSymbol(&loadStoreValue_double, HIP_SYMBOL(load_store_double), LIST_SIZE * sizeof(double), j*sizeof(double), hipMemcpyDeviceToHost);
    
        for(long long i=0; i < LIST_SIZE && loadStoreIndex_double[i] != 0; i++)
        {
           fprintf(profileFile, "%u %.40f\n", loadStoreIndex_double[i], loadStoreValue_double[i]);   
        }
    }
    
    fclose(profileFile);
}