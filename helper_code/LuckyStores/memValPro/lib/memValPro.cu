
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <unistd.h>

#define INDEX
#define LIST_SIZE 5000000
__device__ int index_int = 0;
__device__ int index_float = 0;

__device__ unsigned int load_store_index[LIST_SIZE] = {0};
__device__ unsigned long load_store_value[LIST_SIZE] = {0};
__device__ double load_store_double[LIST_SIZE] = {0.0};
__device__ unsigned int load_store_double_index[LIST_SIZE] = {0};
__device__ unsigned long record_flag;
__device__ unsigned long call_count;

extern "C" __device__ void profileStoreInst(long value, long index){

    if (record_flag == 0)
        return;

    if (index == 166)
    {
        long local_index = atomicAdd(&index_int,1);
        load_store_value[local_index] = (unsigned long)value;
        load_store_index[local_index] = (unsigned int)index;
    }
}

extern "C" __device__ void profileStoreInstfloat(double value, long index){

    if (record_flag == 0)
        return;

    if (index == 166)
    {
        long local_index = atomicAdd(&index_float,1);
        load_store_double[local_index] = value;
        load_store_double_index[local_index] = index;
    } 
}

