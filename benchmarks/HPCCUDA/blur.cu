#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <sys/types.h>
#include <sys/time.h>
#include <math.h>
#include <hip/hip_runtime.h>
#define TX 16
#define TY 32


//#ifdef BAMBOO_PROFILING
//#include "bamboo_profiling.cu"
//#else
#include "record_data.cu"
//#endif

static int const maxlen = 200, rowsize = 521, colsize = 428, linelen = 12;
struct timeval tim;

__global__
void ProcessBlurKernel(int *d_R, int *d_G, int *d_B, int *d_Rnew, int *d_Gnew, int *d_Bnew)
{
	int row = blockIdx.y*blockDim.y+threadIdx.y;
	int col = blockIdx.x*blockDim.x+threadIdx.x;

	int temp = row*colsize+col;
	int temp1 = (row+1)*colsize+col;
	int temp2 = (row-1)*colsize+col;
	int temp3 = row*colsize+(col+1);
	int temp4 = row*colsize+(col-1);



	if(col<colsize && row<rowsize)
	{
		if (row != 0 && row != (rowsize-1) && col != 0 && col != (colsize-1)){
					d_Rnew[temp] = (d_R[temp1]+d_R[temp2]+d_R[temp3]+d_R[temp4])/4;
					d_Gnew[temp] = (d_G[temp1]+d_G[temp2]+d_G[temp3]+d_G[temp4])/4;
					d_Bnew[temp] = (d_B[temp1]+d_B[temp2]+d_B[temp3]+d_B[temp4])/4;
				}
				else if (row == 0 && col != 0 && col != (colsize-1)){
					d_Rnew[temp] = (d_R[temp1]+d_R[temp3]+d_R[temp4])/3;
					d_Gnew[temp] = (d_G[temp1]+d_G[temp3]+d_G[temp4])/3;
					d_Bnew[temp] = (d_B[temp1]+d_B[temp3]+d_B[temp4])/3;
				}
				else if (row == (rowsize-1) && col != 0 && col != (colsize-1)){
					d_Rnew[temp] = (d_R[temp2]+d_R[temp3]+d_R[temp4])/3;
					d_Gnew[temp] = (d_G[temp2]+d_G[temp3]+d_G[temp4])/3;
					d_Bnew[temp] = (d_B[temp2]+d_B[temp3]+d_B[temp4])/3;
				}
				else if (col == 0 && row != 0 && row != (rowsize-1)){
					d_Rnew[temp] = (d_R[temp1]+d_R[temp2]+d_R[temp3])/3;
					d_Gnew[temp] = (d_G[temp1]+d_G[temp2]+d_G[temp3])/3;
					d_Bnew[temp] = (d_B[temp1]+d_B[temp2]+d_B[temp3])/3;
				}
				else if (col == (colsize-1) && row != 0 && row != (rowsize-1)){
					d_Rnew[temp] = (d_R[temp1]+d_R[temp2]+d_R[temp4])/3;
					d_Gnew[temp] = (d_G[temp1]+d_G[temp2]+d_G[temp4])/3;
					d_Bnew[temp] = (d_B[temp1]+d_B[temp2]+d_B[temp4])/3;
				}
				else if (row==0 &&col==0){
					d_Rnew[temp] = (d_R[temp3]+d_R[temp1])/2;
					d_Gnew[temp] = (d_G[temp3]+d_G[temp1])/2;
					d_Bnew[temp] = (d_B[temp3]+d_B[temp1])/2;
				}
				else if (row==0 &&col==(colsize-1)){
					d_Rnew[temp] = (d_R[temp4]+d_R[temp1])/2;
					d_Gnew[temp] = (d_G[temp4]+d_G[temp1])/2;
					d_Bnew[temp] = (d_B[temp4]+d_B[temp1])/2;
				}
				else if (row==(rowsize-1) &&col==0){
					d_Rnew[temp] = (d_R[temp3]+d_R[temp2])/2;
					d_Gnew[temp] = (d_G[temp3]+d_G[temp2])/2;
					d_Bnew[temp] = (d_B[temp3]+d_B[temp2])/2;
				}
				else if (row==(rowsize-1) &&col==(colsize-1)){
					d_Rnew[temp] = (d_R[temp4]+d_R[temp2])/2;
					d_Gnew[temp] = (d_G[temp4]+d_G[temp2])/2;
					d_Bnew[temp] = (d_B[temp4]+d_B[temp2])/2;
				}	
	}
}

__global__
void doCopyKernel(int *d_R, int *d_G, int *d_B, int *d_Rnew, int *d_Gnew, int *d_Bnew)
{
	int row = blockIdx.y*blockDim.y+threadIdx.y;
	int col = blockIdx.x*blockDim.x+threadIdx.x;
	int temp = row*colsize+col;
	if(col<colsize && row<rowsize)
	{
		d_R[temp] = d_Rnew[temp];
		d_G[temp] = d_Gnew[temp];
		d_B[temp] = d_Bnew[temp];
	}
}
void ProcessBlur(int R[rowsize][colsize], int G[rowsize][colsize], int B[rowsize][colsize], int Rnew[rowsize][colsize], int Gnew[rowsize][colsize], int Bnew[rowsize][colsize], int nblurs)
{


	int *d_R, *d_G, *d_B, *d_Rnew, *d_Gnew, *d_Bnew;

	int k;
	int sizea =sizeof(int)*rowsize*colsize;
	
	gettimeofday(&tim, NULL);
	double t5=tim.tv_sec+(tim.tv_usec/1000000.0);

	hipMalloc((void **)&d_R,sizea);
	hipMalloc((void **)&d_G,sizea);
	hipMalloc((void **)&d_B,sizea);
	hipMalloc((void **)&d_Rnew,sizea);
	hipMalloc((void **)&d_Gnew,sizea);
	hipMalloc((void **)&d_Bnew,sizea);

	gettimeofday(&tim, NULL);
	double t6=tim.tv_sec+(tim.tv_usec/1000000.0);
	printf("Allocation of device memory: %.6lf seconds elapsed\n", t6-t5);

	gettimeofday(&tim, NULL);
	double t7=tim.tv_sec+(tim.tv_usec/1000000.0);
	
	hipMemcpy(d_R,R,sizea,hipMemcpyHostToDevice);
	hipMemcpy(d_G,G,sizea,hipMemcpyHostToDevice);
	hipMemcpy(d_B,B,sizea,hipMemcpyHostToDevice);
	dim3 dimGrid(ceil(rowsize/(float)TX),ceil(colsize/(float)TY),1);
	dim3 dimBlock(TX,TY,1);
	
	gettimeofday(&tim, NULL);
	double t1=tim.tv_sec+(tim.tv_usec/1000000.0);

	for(k=0;k<nblurs;k++){

		 bambooLogKernelBegin(k);
		 ProcessBlurKernel<<<dimGrid,dimBlock>>>(d_R,d_G,d_B,d_Rnew,d_Gnew,d_Bnew);
		 bambooLogRecordOff();

		 //bambooLogKernelBegin(1);
		 doCopyKernel<<<dimGrid,dimBlock>>>(d_R,d_G,d_B,d_Rnew,d_Gnew,d_Bnew);
		 //bambooLogKernelEnd(1);
	}

	bambooLogKernelEnd();
	
	hipDeviceSynchronize();

	gettimeofday(&tim, NULL);
	double t2=tim.tv_sec+(tim.tv_usec/1000000.0);
	printf("Doing the blurring: %.6lf seconds elapsed\n", t2-t1);
	    

	    hipMemcpy(R,d_Rnew,sizea,hipMemcpyDeviceToHost);
	    hipMemcpy(G,d_Gnew,sizea,hipMemcpyDeviceToHost);
	    hipMemcpy(B,d_Bnew,sizea,hipMemcpyDeviceToHost);

	gettimeofday(&tim, NULL);
	double t8=tim.tv_sec+(tim.tv_usec/1000000.0);
	printf("Transfer data: %.6lf seconds elapsed\n", t8-(t2-t1)-t7);

	    hipFree(d_R);
		hipFree(d_G);
		hipFree(d_B);
		hipFree(d_Rnew);
		hipFree(d_Gnew);
		hipFree(d_Bnew);
	
}

int main (int argc, const char * argv[]) {
	char str[maxlen], lines[5][maxlen];
	FILE *fp, *fout;
	int nlines = 0;
	unsigned int h1, h2, h3;
	char *sptr;
	int R[rowsize][colsize], G[rowsize][colsize], B[rowsize][colsize];
	int Rnew[rowsize][colsize], Gnew[rowsize][colsize], Bnew[rowsize][colsize];
	int row = 0, col = 0, nblurs, lineno=0;
	nblurs = 10;
    printf("\nGive the number of times to blur the image\n");
    //int icheck = scanf ("%d", &nblurs);
	nblurs = 1;
	
	gettimeofday(&tim, NULL);
	double t3=tim.tv_sec+(tim.tv_usec/1000000.0);
	
	fp = fopen("David.ps", "r");
 
	while(! feof(fp))
	{
		fscanf(fp, "\n%[^\n]", str);
		if (nlines < 5) {strcpy((char *)lines[nlines++],(char *)str);}
		else{
			for (sptr=&str[0];*sptr != '\0';sptr+=6){
				sscanf(sptr,"%2x",&h1);
				sscanf(sptr+2,"%2x",&h2);
				sscanf(sptr+4,"%2x",&h3);
				
				if (col==colsize){
					col = 0;
					row++;
				}
				if (row < rowsize) {
					R[row][col] = h1;
					G[row][col] = h2;
					B[row][col] = h3;
				}
				col++;
			}
		}
	}
	fclose(fp);
	
	gettimeofday(&tim, NULL);
	double t4=tim.tv_sec+(tim.tv_usec/1000000.0);
	printf("Reading input file: %.6lf seconds elapsed\n", t4-t3);


	ProcessBlur(R,G,B,Rnew,Gnew,Bnew,nblurs);//blur the picture


    gettimeofday(&tim, NULL);
	double t9=tim.tv_sec+(tim.tv_usec/1000000.0);

	fout= fopen("DavidBlur.ps", "w");
	int k=0;
	for (k=0;k<nlines;k++) fprintf(fout,"\n%s", lines[k]);
	fprintf(fout,"\n");
	for(row=0;row<rowsize;row++){
		for (col=0;col<colsize;col++){
			fprintf(fout,"%02x%02x%02x",R[row][col],G[row][col],B[row][col]);
			lineno++;
			if (lineno==linelen){
				fprintf(fout,"\n");
				lineno = 0;
			}
		}
	}
	fclose(fout);
	gettimeofday(&tim, NULL);
	double t10=tim.tv_sec+(tim.tv_usec/1000000.0);
	printf("Outputting: %.6lf seconds elapsed\n", t10-t9);

    return 0;
}
