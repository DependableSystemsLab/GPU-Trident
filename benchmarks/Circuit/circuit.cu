#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
//#include "def.cuh"
#include <string.h>


#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>

#ifdef BAMBOO_PROFILING
#include "record_data.cu"
#endif

__global__ void cudaSolve(double * data, double * odata);
inline void findErr(const char * filename, const int line_number);


__device__ int N = 100;
__device__ int THREADS_PER_BLOCK_X = 10;
__device__ int THREADS_PER_BLOCK_Y = 10;

int dynamic_count = 0;

int main(int argc, char **argv){

    int show_solution;
    double tol;

    if(argc == 1){
        show_solution = 0;
        tol = 1e-4;
        printf("Usage: ./q2 tol show_solution.\n");
    }  
    else{
        tol = atof(argv[1]);
        show_solution = atoi(argv[2]);
    }

    double * h_a = 0;
    double * h_a_p = 0;
    double * d_a = 0;
    double * d_a_p = 0;

    int num_elements = 100*100;
    int num_bytes = num_elements * sizeof(double);

    h_a = (double*) calloc(num_elements, sizeof(double));
    h_a_p = (double*) calloc(num_elements, sizeof(double));
    hipMalloc((void **) &d_a, num_bytes);
    hipMemset(d_a, 0, num_bytes);
    hipMalloc((void **) &d_a_p, num_bytes);
    hipMemset(d_a_p, 0, num_bytes);

    // JUSTIN: for ck
    double* h_a_ck = (double*) calloc(num_elements, sizeof(double)); //RM
    double* h_a_p_ck = (double*) calloc(num_elements, sizeof(double)); //TM
    /////////////////////


    double coef[7];
    coef[0] = 0.000001997687531;
    coef[1] = -0.000222144671839;
    coef[2] = 0.009855558615866;
    coef[3] = -0.225735238852272;
    coef[4] = 3.855137564722874;
    coef[5] = -8.706331489366162;
    coef[6] = 33.201397146850610;
    double maxite = 0;
    for(int k = 0;k<7;++k){
        maxite += coef[k];
        if(k!=6)
            maxite *= 100;
    }
    int maxiter = (int) maxite;

    dim3 dimBlock(12, 12);
    dim3 dimGrid(100/dimBlock.x, 100/dimBlock.y);
    int iter;
    double * tmp = 0 ;
    double etime = 0;  
    double resI2 = 0;

    int N = 100;
    //  GpuTimer timer;
    //  timer.Start();
    for(iter=0; iter<maxiter; iter++){
        
        bambooLogKernelBegin(dynamic_count);
        cudaSolve<<<dimGrid, dimBlock>>>(d_a_p, d_a);
        dynamic_count++;
        bambooLogRecordOff();

        if(iter%10==0){
            // timer.Stop();
            //     etime += timer.Elapsed();
            hipMemcpy(h_a, d_a, num_bytes, hipMemcpyDeviceToHost);
            hipMemcpy(h_a_p, d_a_p, num_bytes, hipMemcpyDeviceToHost);  
            resI2 = 0;
            for(int i = 0; i<N; ++i){
                for(int j = 0; j<N; ++j){
                    resI2 += (h_a_p[i*N+j]-h_a[i*N+j])*(h_a_p[i*N+j]-h_a[i*N+j]);
                }
            }
            if(resI2 < tol*tol){
                break;
            }
            // timer.Start();
        }
        tmp = d_a_p;
        d_a_p = d_a;
        d_a = tmp;
    }

    bambooLogKernelEnd();

    //timer.Stop();
    //  etime += timer.Elapsed();

    findErr(__FILE__, __LINE__);
    hipMemcpy(h_a, d_a, num_bytes, hipMemcpyDeviceToHost);
    findErr(__FILE__, __LINE__);
    
    printf("Time elapsed = %g ms \n", etime);
    printf("Top right current = %lg\n", h_a[N*N-1]);
    printf("Iteration = %d\n", iter);
    printf("Bandwidth = %g GB/s\n", (N*N*sizeof(double)*2*iter/1e9)/(etime/1e3));

    if(show_solution == 1){
        FILE *fpo_result = fopen("result.txt","a");

        for(int i = 0; i<N;++i){
            for(int j = 0; j<N;++j){
                //printf("%f ", h_a[i*N+j]);
                fprintf(fpo_result, "%f ", h_a[i*N+j]);
            }
            //printf("\n");
            fprintf(fpo_result, "\n");
        }
        fclose(fpo_result);
    }

    free(h_a);
    free(h_a_p);
    hipFree(d_a);
    hipFree(d_a_p);

}

inline void findErr(const char * filename, const int line_number){

    //#ifdef DEBUG
    hipDeviceSynchronize();
    hipError_t error = hipGetLastError();
    if(error != hipSuccess){
        printf("CUDA error at %s:%i: %s\n", filename, line_number, hipGetErrorString(error));
        exit(-1);
    }
    //#endif

}

__global__ void cudaSolve(double * data, double * odata)
{

  int i = blockDim.y * blockIdx.y + threadIdx.y;
  int j = blockDim.x * blockIdx.x + threadIdx.x;

  int li = threadIdx.y;
  int lj = threadIdx.x;

  int e_li = li + 1;
  int e_lj = lj + 1;

  __shared__ float sdata[12][12];

  unsigned int index = i*N + j;

  if(li < 1){
    if(blockIdx.y > 0){
      sdata[li][e_lj] = data[index - N];
    }
    else{
      sdata[li][e_lj] = 0;
    }

    if(blockIdx.y < (gridDim.y - 1)){
      sdata[e_li + THREADS_PER_BLOCK_Y][e_lj] = data[index + THREADS_PER_BLOCK_Y * N];
    }
    else{
      sdata[e_li + THREADS_PER_BLOCK_Y][e_lj] = 0;
    }
  }

  if(lj < 1){
    if(blockIdx.x > 0){
      sdata[e_li][lj] = data[index - 1];
    }
    else{
      sdata[e_li][lj] = 0;
    }

    if(blockIdx.x < (gridDim.x - 1)){
      sdata[e_li][e_lj + THREADS_PER_BLOCK_X] = data[index + THREADS_PER_BLOCK_X];
    }
    else{
      sdata[e_li][e_lj + THREADS_PER_BLOCK_X] = 0;
    }
  }

  sdata[e_li][e_lj] = data[index];

  double V, invD;

  if(i==0 && j==0){
    V = 1;
    invD = 1./3;
  }
  else{
    V = 0;
    invD = 1./4;
  }

  __syncthreads();

  odata[index] = invD*(V - sdata[e_li-1][e_lj] - sdata[e_li+1][e_lj]
               - sdata[e_li][e_lj-1] - sdata[e_li][e_lj+1]);

}