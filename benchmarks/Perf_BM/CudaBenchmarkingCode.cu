#include "hip/hip_runtime.h"
#include <cstdlib>

#include<iostream>

#include<hip/hip_runtime.h>

#include <sys/time.h>
 //#include <hip/hip_fp16.h>
#include <stdio.h>

#include <stdlib.h>

#include <string.h>

#include <unistd.h>
 //#include <hip/hip_fp16.h>

//#define cudaCores 3584

#include "record_data.cu"

using namespace std;

FILE * fp;

int smCount, totalThreads;

//__float2half
/*void getGPUConfig(){

  hipDeviceProp_t cudaProg;
  hipGetDeviceProperties(&cudaProg,0);
  int SMCount=cudaProg.multiProcessorCount;
  int threadPerBlock=cudaProg.maxThreadsPerBlock;
  int maxThreads=SMCount*threadPerBlock;
  int numberOfBlolcks=__gcd(maxThreads,threadPerBlock);
  int numberOfThreads=maxThreads/numberOfBlolcks;
  cout << "number of blocks:"<< numberOfBlolcks<< endl;
  cout << "number of threads:"<< numberOfThreads<< endl;

}*/

__global__ void multiplyInt(int * a, int * b, int n) {

    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < n) {

        b[i] = b[i] + a[i];

    }
}

__global__ void multiplyFloat(float * a, float * b, int n) {

    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < n) {
        b[i] = b[i] + a[i];
    }
}

/*
__global__ void multiplyHalfFloat(half *a,half *b,int n)
{
  
    int i=blockIdx.x*blockDim.x+threadIdx.x;

	if(i<n){

     b[i]=__float2half(__half2float(a[i])+__half2float(a[i]));

    }
}
*/

int getSPcores(hipDeviceProp_t devProp) {
    int cores = 0;
    int mp = devProp.multiProcessorCount;
    switch (devProp.major) {
    case 2: // Fermi
        if (devProp.minor == 1) cores = mp * 48;
        else cores = mp * 32;
        break;
    case 3: // Kepler
        cores = mp * 192;
        break;
    case 5: // Maxwell
        cores = mp * 128;
        break;
    case 6: // Pascal
        if ((devProp.minor == 1) || (devProp.minor == 2)) cores = mp * 128;
        else if (devProp.minor == 0) cores = mp * 64;
        else printf("Unknown device type\n");
        break;
    case 7: // Volta and Turing
        if ((devProp.minor == 0) || (devProp.minor == 5)) cores = mp * 64;
        else printf("Unknown device type\n");
        break;
    default:
        printf("Unknown device type\n");
        break;
    }
    return cores;
}

int main() {

    //fp= fopen( "GPULogs.txt", "ab" );
    hipDeviceProp_t prop;
    hipGetDeviceProperties( & prop, 0);

    int cudaCores = getSPcores(prop);
    cout << "Cuda Cores:" << cudaCores << endl;
    cout << "Device Name:" << prop.name << endl;
    //fprintf(fp,"\nDeviceName:%s",prop.name);
    cout << "Max Threads Per Block:" << prop.maxThreadsPerBlock << endl;
    //fprintf(fp,"\nMax Threads Per Block:%d",prop.maxThreadsPerBlock);
    smCount = prop.multiProcessorCount;
    cout << "SM Count is:" << smCount << endl;
    //fprintf(fp,"\nSM Count:%d",smCount);
    cout << "Warp Size:" << prop.warpSize << endl;
    //fprintf(fp,"\nWarp Size:%d",prop.warpSize);
    cout << "Clock Rate:" << prop.clockRate << endl;
    //fprintf(fp,"\nClock Rate:%d",prop.clockRate);
    totalThreads = smCount * cudaCores;
    cout << "Total Number of Threads:" << totalThreads << endl;

    int SIZE = totalThreads;

    int * a, * b;
    int * d_a, * d_b;
    float * a_f, * b_f;
    float * d_a_f, * d_b_f;
    //float *a_half,*b_half;
    //half *d_a_half,*d_b_half;

    struct timeval start_int, end_int;
    struct timeval start_float, end_float;
    //struct timeval start_half_float, end_half_float;

    a = new int[SIZE];
    b = new int[SIZE];
    a_f = new float[SIZE];
    b_f = new float[SIZE];
    //a_half=new float[SIZE];
    //b_half=new float[SIZE];

    hipMalloc( & d_a, SIZE * sizeof(int));
    hipMalloc( & d_b, SIZE * sizeof(int));
    hipMalloc( & d_a_f, SIZE * sizeof(float));
    hipMalloc( & d_b_f, SIZE * sizeof(float));
    //hipMalloc(&d_a_half, SIZE*sizeof(half));
    //hipMalloc(&d_b_half, SIZE*sizeof(half));

    int i;

    for (i = 0; i < SIZE; i++) {
        a[i] = i;
        b[i] = 1;
    }

    for (i = 0; i < SIZE; i++) {
        a_f[i] = i + 0.5;
        b_f[i] = i + 1.5;
    }

    /*
        for (i= 0;i< SIZE;i++) {
            a_half[i] = i+1.05;
            b_half[i] = i+2.05;
        }
    */
    hipMemcpy(d_a, a, SIZE * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, SIZE * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_a_f, a_f, SIZE * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b_f, b_f, SIZE * sizeof(float), hipMemcpyHostToDevice);
    //hipMemcpy(d_a_half,a_half,SIZE*sizeof(half), hipMemcpyHostToDevice);
    //hipMemcpy(d_b_half,b_half,SIZE*sizeof(half), hipMemcpyHostToDevice);

    gettimeofday( & start_int, NULL);

    for (i = 0; i < 1000; i++) {

        bambooLogKernelBegin(i);
        multiplyInt << < smCount, cudaCores >>> (d_a, d_b, SIZE);
        bambooLogRecordOff();
    }

    bambooLogKernelEnd();

    gettimeofday( & end_int, NULL);

    hipMemcpy(b, d_b, SIZE * sizeof(int), hipMemcpyDeviceToHost);

    gettimeofday( & start_float, NULL);

    for (i = 0; i < 1000; i++) {

        multiplyFloat << < smCount, cudaCores >>> (d_a_f, d_b_f, SIZE);
    }

    gettimeofday( & end_float, NULL);

    hipMemcpy(b_f, d_b_f, SIZE * sizeof(float), hipMemcpyDeviceToHost);

    //gettimeofday(&start_half_float, NULL);

    //for(i=0;i<1000;i++){

    //  multiplyHalfFloat<<<smCount,cudaCores>>>(d_a_half,d_b_half,SIZE); 

    //}

    //gettimeofday(&end_half_float, NULL); 

    //hipMemcpy(b_half, d_b_half, SIZE*sizeof(float),hipMemcpyDeviceToHost);

    float IOPS = ((SIZE * 1000 * cudaCores) / ((1000.0 * (end_int.tv_sec - start_int.tv_sec) + (end_int.tv_usec - start_int.tv_usec) / 1000.0) / 1000) / 1e9);

    cout << "IOPS:" << IOPS << endl;

    float FLOPS = ((SIZE * 1000 * cudaCores) / ((1000.0 * (end_float.tv_sec - start_float.tv_sec) + (end_float.tv_usec - start_float.tv_usec) / 1000.0) / 1000) / 1e9);

    cout << "GFLOPS:" << FLOPS << endl;

    //float GHOPS = ((SIZE*1000*cudaCores)/ ((1000.0 * (end_half_float.tv_sec - start_half_float.tv_sec) + (end_half_float.tv_usec - start_half_float.tv_usec) / 1000.0)/1000)/1e9);

    //cout << "GHOPS:"<< GHOPS << endl;

    //fprintf(fp,"\nGFLOPS for %s:%f",prop.name,FLOPS);
    //fprintf(fp,"\nIOPS for %s:%f",prop.name,IOPS);
    //fprintf(fp,"\nGHOPS for %s:%f",prop.name,GHOPS);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_a_f);
    hipFree(d_b_f);
    //hipFree(d_b_half);
    //hipFree(d_a_half);

    return 0;
}